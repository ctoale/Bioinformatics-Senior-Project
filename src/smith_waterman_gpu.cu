#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <ctype.h>
#include <math.h>

#include "defs.h"
#include "param.h"
#include "dropgsw2.h"


// inline void gpu_handle_error( hipError_t err, const char* file, int line, int abort = 1 )
// {
// 	if (err != hipSuccess)
// 	{
// 		fprintf (stderr, "gpu error %s, %s, %d\n", hipGetErrorString (err), file, line);
// 		if (abort)
// 			exit (EXIT_FAILURE);
// 	}
// }
// #define gpu_err_chk(e) {gpu_handle_error( e, __FILE__, __LINE__ );}


// kernel function
// __global__
// void smith_waterman_kernel (const unsigned char *aal)
// {
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// }

extern "C" {
  #include "smith_waterman_gpu.h"
}

struct swstr {int H, E;};

extern "C"
int smith_waterman( const unsigned char *aa0,
                    const unsigned char *aa1,
                    int n0,
                    int n1,
                    int low,
                    int up,
                    int **W,
                    int GG,
                    int HH,
                    int MW,
                    struct f_struct *f_str)   //defined in dropgsw.h
{
  // hipError_t err;
  // const unsigned char *d_aa1;
  // const unsigned char *d_aa1p;
  const unsigned char *aa1p;
  // struct f_struct *d_f_str;
  // register int *d_pwaa;
  register int *pwaa;
  // register struct swstr *d_ssj;
  register struct swstr *ssj;
  // struct swstr *d_ss;
  struct swstr *ss;
  register int h, e, f, p;
  int temp, score;
  int gap_ext, n_gap_init;
  // int *d_score_pt;
  int *score_pt;

  ss = f_str->ss;
  ss[n0].H = -1;
  ss[n0].E = 1;

  n_gap_init = GG + HH;
  gap_ext = -HH;	/* GG, HH are both positive,
                gap_ext penalty should be negative */

  score = 0;
  for (h=0; h<n0; h++) {	  /* initialize 0th row */
    ss[h].H = ss[h].E = 0;

  }
  
  aa1p=aa1;

  // err = hipMalloc ((void**) &d_aa1, n1 * sizeof(char));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_aa1p, n1 * sizeof(char));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_f_str, sizeof(f_struct));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_pwaa, sizeof(int));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_ssj, sizeof(swstr));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_ss, sizeof(swstr));
  // gpu_err_chk(err);
  // err = hipMalloc ((void**) &d_score_pt, sizeof(int));
  // gpu_err_chk(err);

  // err = hipMemcpy (d_aa1, h_aa1, n1 * sizeof(char), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_aa1p, aa1p, n1 * sizeof(char), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_f_str, f_str, sizeof(f_struct), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_pwaa, pwaa, sizeof(int), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_ssj, ssj, sizeof(swstr), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_ss, ss, sizeof(swstr), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  // err = hipMemcpy (d_score_pt, score_pt, sizeof(int), hipMemcpyHostToDevice);
  // gpu_err_chk(err);
  
  while (*aa1p) {		/* relies on d_aa1[n1]==0 for EOS flag */
    /* waa_s has the offsets for each residue in d_aa0 into pam2
  */
    /* waa_s has complexity (-S) dependent scores */
    pwaa = f_str->waa_s + (*aa1p++)*n0;
    ssj = ss;

    e = f = h = p = 0;
  zero_f:	/* in this section left-gap f==0, and is never examined */

    while (1) {	/* build until h > n_gap_init (f < 0 until h > n_gap_init) */
              /* bump through the pam[][]'s for each of the d_aa1[] matches to
              d_aa0[], because of the way *d_pwaa is set up */


      h = p + *pwaa++;		/* increment diag value */
      p = ssj->H;		/* get next diag value */
      if ((e = ssj->E) > 0 ) {	/* >0 from up-gap */
    if (p == -1) goto next_row;	/* done, -1=d_ss[n0].H sentinel */
    if (h < e) h = e;	/* up-gap better than diag */
    else 
      if (h > n_gap_init) {	/* we won't starting a new up-gap */
        e += gap_ext;	/* but we might be extending one */
        goto transition;	/* good h > n_gap_diag; scan f */
      }
    e += gap_ext;		/* up-gap decreased */
    ssj->E =  (e > 0) ?  e : 0;	/* set to 0 if < 0 */
    ssj++->H = h;		/* diag match updated */
      }
      else {			/* up-gap (->E) is 0 */

    if ( h > 0) {		/* diag > 0 */
      if (h > n_gap_init) {	/* we won't be starting a new up-gap */
        e = 0;		/* and we won't be extending one */
        goto transition;	/* good h > n_gap_diag; scan f */
      }
      ssj++->H = h;		/* update diag */
    }
    else ssj++->H = 0;	/* update diag to 0 */
      }
    }

    /* here h > n_gap_init and h > e, => the next f will be > 0 */
  transition:

#ifdef DEBUG
    if ( h > 10000) 
      fprintf(stderr,"h: %d d_ssj: %d\n",h, (int)(ssj-ss));
#endif
    if ( score < h ) score = h;	/* save best score, only when h > n_gap_init */

    temp = h - n_gap_init;	/* best score for starting a new gap */
    if ( f < temp ) f = temp;	/* start a left-gap? */
    if ( e < temp ) e = temp;	/* start an up-gap? */
    ssj->E = ( e > 0 ) ? e : 0;	/* update up-gap */

    ssj++->H = h;		/* update diag */
    e = 0;

    do {			/* stay here until f <= 0 */
      h = p + *pwaa++;		/* diag + match/mismatch */
      p = ssj->H;		/* save next (right) diag */

      if ( h < f ) h = f;	/* update diag using left gap */
      f += gap_ext;		/* update next left-gap */

      if ((e = ssj->E) > 0) {	/* good up gap */
    if (p == -1) goto next_row;	/* at the end of the row */
    if ( h < e ) h = e;	/* update diag using up-gap */
    else
      if ( h > n_gap_init ) {
        e += gap_ext;	/* update up gap */
        goto transition;	/* good diag > n_gap_init, restart */
      }
    e += gap_ext;		/* update up-gap */
    ssj->E = (e > 0) ? e : 0;	/* e must be >= 0 */
    ssj++->H = h;		/* update diag */
      }
      else {			/* up-gap <= 0 */
    if ( h > n_gap_init ) {
      e = 0;
      goto transition;	/* good diag > n_gap_init; restart */
    }
    ssj++->H = h;		/* update diag */
      }
    } while ( f > 0 );		/* while left gap f > 0  */
    goto zero_f;		/* otherwise, go to f==0 section */
  next_row:
    ;
  }		/* end while(*aap1) {} */

  return score;  
}



